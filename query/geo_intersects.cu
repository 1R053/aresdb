//  Copyright (c) 2017-2018 Uber Technologies, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <thrust/iterator/discard_iterator.h>
#include <thrust/transform.h>
#include <algorithm>
#include <vector>
#include "query/algorithm.h"
#include "query/binder.h"

namespace ares {
class GeoContext {
 protected:
  GeoContext(GeoShapeBatch geoShapes,
             int indexVectorLength,
             uint32_t startCount,
             uint32_t *outputPredicate,
             void *cudaStream)
      : geoShapes(geoShapes),
        indexVectorLength(indexVectorLength),
        startCount(startCount), outputPredicate(outputPredicate),
        cudaStream(reinterpret_cast<hipStream_t>(cudaStream)) {}
  GeoShapeBatch geoShapes;
  int indexVectorLength;
  uint32_t startCount;
  uint32_t *outputPredicate;
  hipStream_t cudaStream;
 public:
  hipStream_t getStream() const {
    return cudaStream;
  }
};

class GeoIntersectionContext : public GeoContext {
 public:
  GeoIntersectionContext(GeoShapeBatch geoShapes,
                         int indexVectorLength,
                         uint32_t startCount,
                         RecordID **recordIDVectors,
                         int numForeignTables,
                         uint32_t *outputPredicate,
                         bool inOrOut,
                         void *cudaStream) : GeoContext(geoShapes,
                                                        indexVectorLength,
                                                        startCount,
                                                        outputPredicate,
                                                        cudaStream),
                                             foreignTableRecordIDVectors(
                                                 recordIDVectors),
                                             numForeignTables(numForeignTables),
                                             inOrOut(inOrOut) {}
  template<typename InputIterator>
  int run(uint32_t *indexVector, InputIterator inputIterator);

 private:
  RecordID **foreignTableRecordIDVectors;
  int numForeignTables;
  bool inOrOut;

  template<typename IndexZipIterator>
  int executeRemoveIf(IndexZipIterator indexZipIterator);
};

class GeoIntersectionJoinContext : public GeoContext {
 public:
  GeoIntersectionJoinContext(GeoShapeBatch geoShapes,
                             DimensionOutputVector dimOut,
                             int indexVectorLength,
                             uint32_t startCount,
                             uint32_t *outputPredicate,
                             void *cudaStream) : GeoContext(geoShapes,
                                                            indexVectorLength,
                                                            startCount,
                                                            outputPredicate,
                                                            cudaStream),
                                                 dimOut(dimOut) {}
  template<typename InputIterator>
  int run(uint32_t *indexVector, InputIterator inputIterator);

 private:
  DimensionOutputVector dimOut;
};

// Base binder class for GeoIntersectionJoinContext and GeoIntersectionContext.
template<typename Context>
class GeoInputVectorBinder : public InputVectorBinderBase<Context, 1, 1> {
  typedef InputVectorBinderBase<Context, 1, 1> super_t;
 protected:
  explicit GeoInputVectorBinder(Context context,
                                std::vector<InputVector> inputVectors,
                                uint32_t *indexVector, uint32_t *baseCounts,
                                uint32_t startCount) : super_t(context,
                                                               inputVectors,
                                                               indexVector,
                                                               baseCounts,
                                                               startCount) {
  }
 public:
  template<typename ...InputIterators>
  int bind(InputIterators... boundInputIterators);
};

// Specialize InputVectorBinder for GeoIntersectionJoinContext.
template<>
class InputVectorBinder<GeoIntersectionJoinContext, 1>
    : public GeoInputVectorBinder<
        GeoIntersectionJoinContext> {
  typedef GeoInputVectorBinder<GeoIntersectionJoinContext> super_t;
 public:
  explicit InputVectorBinder(GeoIntersectionJoinContext context,
                             std::vector<InputVector> inputVectors,
                             uint32_t *indexVector, uint32_t *baseCounts,
                             uint32_t startCount) : super_t(context,
                                                            inputVectors,
                                                            indexVector,
                                                            baseCounts,
                                                            startCount) {
  }
};

// Specialize InputVectorBinder for GeoIntersectionContext.
template<>
class InputVectorBinder<GeoIntersectionContext, 1>
    : public GeoInputVectorBinder<GeoIntersectionContext> {
  typedef GeoInputVectorBinder<GeoIntersectionContext> super_t;
 public:
  explicit InputVectorBinder(GeoIntersectionContext context,
                             std::vector<InputVector> inputVectors,
                             uint32_t *indexVector, uint32_t *baseCounts,
                             uint32_t startCount) : super_t(context,
                                                            inputVectors,
                                                            indexVector,
                                                            baseCounts,
                                                            startCount) {
  }
};

}  // namespace ares

CGoCallResHandle GeoBatchIntersects(
    GeoShapeBatch geoShapes, InputVector points, uint32_t *indexVector,
    int indexVectorLength, uint32_t startCount, RecordID **recordIDVectors,
    int numForeignTables, uint32_t *outputPredicate, bool inOrOut,
    void *cudaStream, int device) {
  CGoCallResHandle resHandle = {0, nullptr};
  try {
#ifdef RUN_ON_DEVICE
    hipSetDevice(device);
#endif
    ares::GeoIntersectionContext
        ctx(geoShapes, indexVectorLength, startCount,
            recordIDVectors, numForeignTables, outputPredicate, inOrOut,
            cudaStream);
    std::vector<InputVector> inputVectors = {points};
    ares::InputVectorBinder<ares::GeoIntersectionContext, 1>
        binder(ctx, inputVectors, indexVector, nullptr, startCount);
    resHandle.res = reinterpret_cast<void *>(binder.bind());
    CheckCUDAError("GeoBatchIntersects");
    return resHandle;
  } catch (const std::exception &e) {
    std::cerr << "Exception happened when doing GeoBatchIntersects:" << e.what()
              << std::endl;
    resHandle.pStrErr = strdup(e.what());
  }
  return resHandle;
}

CGoCallResHandle GeoBatchIntersectsJoin(
    GeoShapeBatch geoShapes, DimensionOutputVector dimOut,
    InputVector points, uint32_t *indexVector, int indexVectorLength,
    uint32_t startCount, uint32_t *outputPredicate, void *cudaStream,
    int device) {
  CGoCallResHandle resHandle = {nullptr, nullptr};
  try {
#ifdef RUN_ON_DEVICE
    hipSetDevice(device);
#endif
    ares::GeoIntersectionJoinContext
        ctx(geoShapes, dimOut, indexVectorLength, startCount,
            outputPredicate, cudaStream);
    std::vector<InputVector> inputVectors = {points};
    ares::InputVectorBinder<ares::GeoIntersectionJoinContext, 1>
        binder(ctx, inputVectors, indexVector, nullptr, startCount);
    resHandle.res = reinterpret_cast<void *>(binder.bind());
    CheckCUDAError("GeoIntersectsJoin");
    return resHandle;
  } catch (const std::exception &e) {
    std::cerr << "Exception happened when doing GeoIntersectsJoin:" << e.what()
              << std::endl;
    resHandle.pStrErr = strdup(e.what());
  }
  return resHandle;
}

namespace ares {

template<typename Context>
template<typename ...InputIterators>
int GeoInputVectorBinder<Context>::bind(
    InputIterators... boundInputIterators) {
  InputVector input = super_t::inputVectors[0];
  uint32_t *indexVector = super_t::indexVector;
  uint32_t startCount = super_t::startCount;
  Context context = super_t::context;

  if (input.Type == VectorPartyInput) {
    VectorPartySlice points = input.Vector.VP;
    if (points.DataType != GeoPoint) {
      throw std::invalid_argument(
          "only geo point column are allowed in geo_intersects");
    }

    if (points.BasePtr == nullptr) {
      return 0;
    }

    uint8_t *basePtr = points.BasePtr;
    uint32_t nullsOffset = points.NullsOffset;
    uint32_t valueOffset = points.ValuesOffset;
    uint8_t startingIndex = points.StartingIndex;
    uint8_t stepInBytes = 8;
    uint32_t length = points.Length;
    auto columnIter = make_column_iterator<GeoPointT>(
        indexVector, nullptr, startCount, basePtr, nullsOffset, valueOffset,
        length, stepInBytes, startingIndex);
    return context.run(indexVector, columnIter);
  } else if (input.Type == ForeignColumnInput) {
    DataType dataType = input.Vector.ForeignVP.DataType;

    if (dataType != GeoPoint) {
      throw std::invalid_argument(
          "only geo point column are allowed in geo_intersects");
    }
    // Note: for now foreign vectors are dimension table columns
    // that are not compressed nor pre sliced
    RecordID *recordIDs = input.Vector.ForeignVP.RecordIDs;
    const int32_t numBatches = input.Vector.ForeignVP.NumBatches;
    const int32_t baseBatchID = input.Vector.ForeignVP.BaseBatchID;
    VectorPartySlice *vpSlices = input.Vector.ForeignVP.Batches;
    const int32_t numRecordsInLastBatch =
        input.Vector.ForeignVP.NumRecordsInLastBatch;
    bool hasDefault = input.Vector.ForeignVP.DefaultValue.HasDefault;
    DefaultValue defaultValueStruct = input.Vector.ForeignVP.DefaultValue;
    uint8_t stepInBytes = getStepInBytes(dataType);

    ForeignTableIterator<GeoPointT> *vpIters = prepareForeignTableIterators(
        numBatches,
        vpSlices,
        stepInBytes,
        hasDefault,
        defaultValueStruct.Value.GeoPointVal,
        context.getStream());
    int res =
        context.run(indexVector, RecordIDJoinIterator<GeoPointT>(
            recordIDs,
            numBatches,
            baseBatchID,
            vpIters,
            numRecordsInLastBatch,
            nullptr, 0));
    release(vpIters);
    return res;
  }
  throw std::invalid_argument(
      "Unsupported data type " + std::to_string(__LINE__)
          + "for geo intersection contexts");
}

// GeoRemoveFilter
template<typename Value>
struct GeoRemoveFilter {
  explicit GeoRemoveFilter(GeoPredicateIterator predicates, bool inOrOut)
      : predicates(predicates), inOrOut(inOrOut) {}

  GeoPredicateIterator predicates;
  bool inOrOut;

  __host__ __device__
  bool operator()(const Value &index) {
    return inOrOut == predicates[thrust::get<0>(index)] < 0;
  }
};

// actual function for executing geo filter in batch.
template<typename IndexZipIterator>
int GeoIntersectionContext::executeRemoveIf(IndexZipIterator indexZipIterator) {
  GeoPredicateIterator predIter(outputPredicate, geoShapes.TotalWords);
  GeoRemoveFilter<typename IndexZipIterator::value_type> removeFilter(predIter,
                                                                      inOrOut);
#ifdef RUN_ON_DEVICE
  return thrust::remove_if(thrust::cuda::par.on(cudaStream), indexZipIterator,
                           indexZipIterator + indexVectorLength, removeFilter) -
         indexZipIterator;
#else
  return thrust::remove_if(thrust::host, indexZipIterator,
                           indexZipIterator + indexVectorLength, removeFilter) -
      indexZipIterator;
#endif
}

// run intersection algorithm for points and 1 geoshape, side effect is
// modifying output predicate vector
template<typename InputIterator>
void calculateBatchIntersection(GeoShapeBatch geoShapes,
                                InputIterator geoPoints, uint32_t *indexVector,
                                int indexVectorLength, uint32_t startCount,
                                uint32_t *outputPredicate, bool inOrOut,
                                hipStream_t cudaStream) {
  auto geoIter = make_geo_batch_intersect_iterator(geoPoints, geoShapes,
                                                   outputPredicate, inOrOut);
  int64_t iterLength = (int64_t) indexVectorLength * geoShapes.TotalNumPoints;

  thrust::for_each(
#ifdef RUN_ON_DEVICE
      thrust::cuda::par.on(reinterpret_cast<hipStream_t>(cudaStream)),
#else
      thrust::host,
#endif
      geoIter, geoIter + iterLength, VoidFunctor());
}

template<typename InputIterator>
int GeoIntersectionContext::run(uint32_t *indexVector,
                                InputIterator inputIterator) {
  calculateBatchIntersection(geoShapes,
                             inputIterator,
                             indexVector,
                             indexVectorLength,
                             startCount,
                             outputPredicate,
                             inOrOut,
                             cudaStream);

  switch (numForeignTables) {
    case 0: {
      IndexZipIteratorMaker<0> maker;
      return executeRemoveIf(maker.make(indexVector,
                                        foreignTableRecordIDVectors));
    }
    case 1: {
      IndexZipIteratorMaker<1> maker;
      return executeRemoveIf(maker.make(indexVector,
                                        foreignTableRecordIDVectors));
    }
    case 2: {
      IndexZipIteratorMaker<2> maker;
      return executeRemoveIf(maker.make(indexVector,
                                        foreignTableRecordIDVectors));
    }
    case 3: {
      IndexZipIteratorMaker<3> maker;
      return executeRemoveIf(maker.make(indexVector,
                                        foreignTableRecordIDVectors));
    }
    case 4: {
      IndexZipIteratorMaker<4> maker;
      return executeRemoveIf(maker.make(indexVector,
                                        foreignTableRecordIDVectors));
    }
    case 5: {
      IndexZipIteratorMaker<5> maker;
      return executeRemoveIf(maker.make(indexVector,
                                        foreignTableRecordIDVectors));
    }
    case 6: {
      IndexZipIteratorMaker<6> maker;
      return executeRemoveIf(maker.make(indexVector,
                                        foreignTableRecordIDVectors));
    }
    case 7: {
      IndexZipIteratorMaker<7> maker;
      return executeRemoveIf(maker.make(indexVector,
                                        foreignTableRecordIDVectors));
    }
    case 8: {
      IndexZipIteratorMaker<8> maker;
      return executeRemoveIf(maker.make(indexVector,
                                        foreignTableRecordIDVectors));
    }
    default:throw std::invalid_argument("only support up to 8 foreign tables");
  }
}

struct is_non_negative {
  __host__ __device__
  bool operator()(const int val) { return val >= 0; }
};

template<typename InputIterator>
int GeoIntersectionJoinContext::run(uint32_t *indexVector,
                                    InputIterator inputIterator) {
  calculateBatchIntersection(geoShapes, inputIterator, indexVector,
                             indexVectorLength, startCount, outputPredicate,
                             true, cudaStream);
  typedef thrust::tuple<int8_t, uint8_t> DimensionOutputIterValue;
  GeoPredicateIterator geoPredicateIter(outputPredicate,
                                        geoShapes.TotalWords);

  auto zippedShapeIndexIter = thrust::make_zip_iterator(thrust::make_tuple(
      geoPredicateIter, thrust::constant_iterator<uint8_t>(1)));

  thrust::transform_if(
#ifdef RUN_ON_DEVICE
      thrust::cuda::par.on(reinterpret_cast<hipStream_t>(cudaStream)),
#else
      thrust::host,
#endif
      zippedShapeIndexIter, zippedShapeIndexIter + indexVectorLength,
      geoPredicateIter,
      ares::make_dimension_output_iterator<uint8_t>(dimOut.DimValues,
                                                    dimOut.DimNulls),
      thrust::identity<DimensionOutputIterValue>(), is_non_negative());
  return 0;
}

}  // namespace ares
