//  Copyright (c) 2017-2018 Uber Technologies, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <cstdio>
#include <cstring>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "../memory.h"

const int MAX_ERROR_LEN = 100;

// checkCUDAError checks the cuda error of last runtime calls and returns the
// pointer to the buffer of error message. This buffer needs to be released
// by caller or upper callers.
char *checkCUDAError(const char *message) {
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    char *buffer = reinterpret_cast<char *>(malloc(MAX_ERROR_LEN));
    snprintf(buffer, MAX_ERROR_LEN,
             "ERROR when calling CUDA functions from host: %s: %s\n",
             message, hipGetErrorString(error));
    return buffer;
  }
  return NULL;
}

extern "C" CGoCallResHandle HostAlloc(size_t bytes) {
  CGoCallResHandle resHandle = {NULL, NULL};
  // hipHostMallocPortable makes sure that the allocation is associated with all
  // devices, not just the current device.
  hipHostAlloc(&resHandle.res, bytes, hipHostMallocPortable);
  memset(resHandle.res, 0, bytes);
  resHandle.pStrErr = checkCUDAError("Allocate");
  return resHandle;
}

extern "C" CGoCallResHandle HostFree(void *p) {
  CGoCallResHandle resHandle = {NULL, NULL};
  hipHostFree(p);
  resHandle.pStrErr = checkCUDAError("Free");
  return resHandle;
}

extern "C" CGoCallResHandle CreateCudaStream(int device) {
  CGoCallResHandle resHandle = {NULL, NULL};
  hipSetDevice(device);
  hipStream_t s = NULL;
  hipStreamCreate(&s);
  resHandle.res = reinterpret_cast<void *>(s);
  resHandle.pStrErr = checkCUDAError("CreateCudaStream");
  return resHandle;
}

extern "C" CGoCallResHandle WaitForCudaStream(void *s, int device) {
  CGoCallResHandle resHandle = {NULL, NULL};
  hipSetDevice(device);
  hipStreamSynchronize((hipStream_t) s);
  resHandle.pStrErr = checkCUDAError("WaitForCudaStream");
  return resHandle;
}

extern "C" CGoCallResHandle DestroyCudaStream(void *s, int device) {
  CGoCallResHandle resHandle = {NULL, NULL};
  hipSetDevice(device);
  hipStreamDestroy((hipStream_t) s);
  resHandle.pStrErr = checkCUDAError("DestroyCudaStream");
  return resHandle;
}

extern "C" CGoCallResHandle DeviceAllocate(size_t bytes, int device) {
  CGoCallResHandle resHandle = {NULL, NULL};
  hipSetDevice(device);
  hipMalloc(&resHandle.res, bytes);
  hipMemset(resHandle.res, 0, bytes);
  resHandle.pStrErr = checkCUDAError("DeviceAllocate");
  return resHandle;
}

extern "C" CGoCallResHandle DeviceFree(void *p, int device) {
  CGoCallResHandle resHandle = {NULL, NULL};
  hipSetDevice(device);
  hipFree(p);
  resHandle.pStrErr = checkCUDAError("DeviceFree");
  return resHandle;
}

extern "C" CGoCallResHandle AsyncCopyHostToDevice(
    void *dst, void *src, size_t bytes, void *stream, int device) {
  CGoCallResHandle resHandle = {NULL, NULL};
  hipSetDevice(device);
  hipMemcpyAsync(dst, src, bytes,
                  hipMemcpyHostToDevice, (hipStream_t) stream);
  resHandle.pStrErr = checkCUDAError("AsyncCopyHostToDevice");
  return resHandle;
}

extern "C" CGoCallResHandle AsyncCopyDeviceToDevice(
    void *dst, void *src, size_t bytes, void *stream, int device) {
  CGoCallResHandle resHandle = {NULL, NULL};
  hipSetDevice(device);
  hipMemcpyAsync(dst, src, bytes,
                  hipMemcpyDeviceToDevice, (hipStream_t) stream);
  resHandle.pStrErr = checkCUDAError("AsyncCopyDeviceToDevice");
  return resHandle;
}

extern "C" CGoCallResHandle AsyncCopyDeviceToHost(
    void *dst, void *src, size_t bytes, void *stream, int device) {
  CGoCallResHandle resHandle = {NULL, NULL};
  hipSetDevice(device);
  hipMemcpyAsync(dst, src, bytes,
                  hipMemcpyDeviceToHost, (hipStream_t) stream);
  resHandle.pStrErr = checkCUDAError("AsyncCopyDeviceToHost");
  return resHandle;
}

extern "C" CGoCallResHandle GetDeviceCount() {
  CGoCallResHandle resHandle = {NULL, NULL};
  hipGetDeviceCount(reinterpret_cast<int *>(&resHandle.res));
  resHandle.pStrErr = checkCUDAError("GetDeviceCount");
  return resHandle;
}

extern "C" CGoCallResHandle GetDeviceGlobalMemoryInMB(int device) {
  CGoCallResHandle resHandle = {NULL, NULL};
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);
  resHandle.res = reinterpret_cast<void *>(prop.totalGlobalMem / (1024 * 1024));
  resHandle.pStrErr = checkCUDAError("GetDeviceGlobalMemoryInMB");
  return resHandle;
}

extern "C" CGoCallResHandle CudaProfilerStart() {
  CGoCallResHandle resHandle = {NULL, NULL};
  hipProfilerStart();
  resHandle.pStrErr = checkCUDAError("hipProfilerStart");
  return resHandle;
}

extern "C" CGoCallResHandle CudaProfilerStop() {
  CGoCallResHandle resHandle = {NULL, NULL};
  hipDeviceSynchronize();
  hipProfilerStop();
  resHandle.pStrErr = checkCUDAError("hipProfilerStop");
  return resHandle;
}
